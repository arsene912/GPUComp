#include <stdio.h>
#include "chTimer.h"

int main(int argc, char *argv[]) {
    int *dmem, *hpage, *hpin;
    int size = 1024;   // 1kB
    chTimerTimestamp start, stop;
    double microseconds;
    
    for (int i=0; i<21; i++) {
        hipMalloc((void**)&dmem, size);   // menory on device
        hpage = (int*) malloc(size);   // pageable memory on host
        hipHostMalloc((void**)&hpin, size);   // pinned memory on host
        
        chTimerGetTime( &start );
        hipMemcpy( dmem, hpage, size, hipMemcpyHostToDevice );
        chTimerGetTime( &stop );
        microseconds = 1e6*chTimerElapsedTime( &start, &stop );
        printf("%d kB; H2D; pageable; %.2f us\n", size/1024, microseconds); fflush(stdout);
        
        chTimerGetTime( &start );
        hipMemcpy( dmem, hpin, size, hipMemcpyHostToDevice );
        chTimerGetTime( &stop );
        microseconds = 1e6*chTimerElapsedTime( &start, &stop );
        printf("%d kB; H2D; pinned; %.2f us\n", size/1024, microseconds); fflush(stdout);
        
        chTimerGetTime( &start );
        hipMemcpy( hpage, dmem, size, hipMemcpyDeviceToHost );
        chTimerGetTime( &stop );
        microseconds = 1e6*chTimerElapsedTime( &start, &stop );
        printf("%d kB; D2H; peagable; %.2f us\n", size/1024, microseconds); fflush(stdout);
        
        chTimerGetTime( &start );
        hipMemcpy( hpin, dmem, size, hipMemcpyDeviceToHost );
        chTimerGetTime( &stop );
        microseconds = 1e6*chTimerElapsedTime( &start, &stop );
        printf("%d kB; D2H; pinned; %.2f us\n", size/1024, microseconds); fflush(stdout);
        
        hipFree(dmem); free(hpage); hipHostFree(hpin);
        size = size*2;   // double the size
    }
}